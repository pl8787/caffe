#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void PReLUForwardShare(const int n, const Dtype* in, Dtype* out,
    const Dtype* negative_slope_share) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] > 0 ? in[index] : in[index] * negative_slope_share[0];
  }
}

template <typename Dtype>
__global__ void PReLUForwardChannel(const int n, const Dtype* in, Dtype* out,
    const Dtype* negative_slope, const int channel_size, const int num_size) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] > 0 ? in[index] : in[index] * negative_slope[(index % num_size) / channel_size];
  }
}

template <typename Dtype>
void PReLULayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    vector<Blob<Dtype>*>* top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = (*top)[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  const int channel_size = bottom[0]->height() * bottom[0]->width();
  const int num_size = bottom[0]->channels() * channel_size;
  const Dtype* negative_slope = this->blobs_[0]->gpu_data();
  
  if (mode_ == PReLUParameter_ReLUMode_SHARE) {
    PReLUForwardShare<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, top_data, negative_slope);
  } else if (mode_ == PReLUParameter_ReLUMode_CWISE) {
    PReLUForwardChannel<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, top_data, negative_slope, channel_size, num_size);
  }
  CUDA_POST_KERNEL_CHECK;
  // << " count: " << count << " bottom_data: "
  //     << (unsigned long)bottom_data
  //     << " top_data: " << (unsigned long)top_data
  //     << " blocks: " << CAFFE_GET_BLOCKS(count)
  //     << " threads: " << CAFFE_CUDA_NUM_THREADS;
}

template <typename Dtype>
__global__ void PReLUBackwardErrorShare(const int n, const Dtype* in_diff,
    const Dtype* in_data, Dtype* out_diff, const Dtype* negative_slope) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = in_diff[index] * ((in_data[index] > 0)
        + (in_data[index] <= 0) * negative_slope[0]);
  }
}

template <typename Dtype>
__global__ void PReLUBackwardErrorChannel(const int n, const Dtype* in_diff,
    const Dtype* in_data, Dtype* out_diff, const Dtype* negative_slope,
    const int channel_size, const int num_size) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = in_diff[index] * ((in_data[index] > 0)
        + (in_data[index] <= 0) * negative_slope[(index % num_size) / channel_size]);
  }
}

template <typename Dtype>
__global__ void PReLUBackwardWeight(const int n, const Dtype* in_diff,
    const Dtype* in_data, Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = in_diff[index] * (in_data[index] <= 0) * in_data[index];
  }
}

template <typename Dtype>
void PReLULayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    vector<Blob<Dtype>*>* bottom) {
  const Dtype* bottom_data = (*bottom)[0]->gpu_data();
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = (*bottom)[0]->mutable_gpu_diff();
  const int count = (*bottom)[0]->count();
  const int channel_size = (*bottom)[0]->height() * (*bottom)[0]->width();
  const int num_size = (*bottom)[0]->channels() * channel_size;
  const Dtype* negative_slope = this->blobs_[0]->gpu_data();

#if 1
  const Dtype* bottom_data_cpu = (*bottom)[0]->cpu_data();
  const Dtype* top_diff_cpu = top[0]->cpu_diff();
  Dtype* weight_diff_cpu = this->blobs_[0]->mutable_cpu_diff();

  if (this->param_propagate_down_[0]) {
    caffe_set(this->blobs_[0]->count(), (Dtype)0, weight_diff_cpu);
    if (mode_ == PReLUParameter_ReLUMode_SHARE) {
      for (int i = 0; i < count; ++i) {
        weight_diff_cpu[0] += top_diff_cpu[i]*(bottom_data_cpu[i] <= 0)*bottom_data_cpu[i];
      }
    } else if (mode_ == PReLUParameter_ReLUMode_CWISE) {
      for (int i = 0; i < top[0]->num(); ++i) {
		for (int j = 0; j < top[0]->channels(); ++j) {
		  for (int k = 0; k < channel_size; ++k) {
            weight_diff_cpu[j] += top_diff_cpu[k]*(bottom_data_cpu[k] <= 0)*bottom_data_cpu[k];
		  }
		  top_diff_cpu += channel_size;
		  bottom_data_cpu += channel_size;
		}
      }
    }
  }
#else
  if (this->param_propagate_down_[0]) {
    Dtype* weight_diff = this->blobs_[0]->mutable_gpu_diff();
    Dtype* weight_diff_mat_data = weight_diff_mat.mutable_gpu_data();

    caffe_gpu_set(this->blobs_[0]->count(), (Dtype)0, weight_diff);
    PReLUBackwardWeight<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
             count, top_diff, bottom_data, weight_diff_mat_data);
    if (mode_ == PReLUParameter_ReLUMode_SHARE) {
	  Dtype* ones_data = ones.mutable_gpu_data();
      caffe_gpu_dot_gpu(count, weight_diff_mat_data, ones_data, weight_diff);
    } else if (mode_ == PReLUParameter_ReLUMode_CWISE) {
	  Dtype* ones_data = ones.mutable_gpu_data();
  	  Dtype* diff_temp = temp_weight_diff.mutable_gpu_data();

      for (int i = 0; i < top[0]->num(); ++i) {
        for (int j = 0; j < top[0]->channels(); ++j) {
          caffe_gpu_dot_gpu(channel_size, weight_diff_mat_data + i*num_size + j*channel_size, ones_data, diff_temp + j);
	    }
		caffe_gpu_add(top[0]->channels(), weight_diff, diff_temp, weight_diff);
      }
    } 
  }
#endif
  if (propagate_down[0]) {
    if (mode_ == PReLUParameter_ReLUMode_SHARE) {
      PReLUBackwardErrorShare<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
          count, top_diff, bottom_data, bottom_diff, negative_slope);
    } else if (mode_ == PReLUParameter_ReLUMode_CWISE) {
      PReLUBackwardErrorChannel<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
          count, top_diff, bottom_data, bottom_diff, negative_slope, channel_size, num_size);
    }
    CUDA_POST_KERNEL_CHECK;
  }
}

INSTANTIATE_CLASS(PReLULayer);


}  // namespace caffe
